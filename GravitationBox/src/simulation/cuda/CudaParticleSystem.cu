#include "hip/hip_runtime.h"
#include "cuda/CudaParticleSystem.h"
#include "cpu/CpuParticleSystem.h"
#include "utils/cuda_helper.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

CudaParticleSystem::CudaParticleSystem(uint32_t count, float radius, ParticleSolver *solver)
	: ParticleSystem(count, radius, solver)
{
	CUDA_CHECK_NR(hipMalloc(&PosX, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&SortedPosX, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&PosY, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&SortedPosY, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&VelX, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&SortedVelX, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&VelY, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&SortedVelY, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&ForceX, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMemset(ForceX, 0, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&SortedForceX, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&ForceY, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMemset(ForceY, 0, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&SortedForceY, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&Mass, Count * sizeof(float)));
	CUDA_CHECK_NR(hipMalloc(&Color, Count * sizeof(glm::vec4)));
}

CudaParticleSystem::~CudaParticleSystem()
{
	CUDA_CHECK_NR(hipFree(PosX));
	CUDA_CHECK_NR(hipFree(SortedPosX));
	CUDA_CHECK_NR(hipFree(PosY));
	CUDA_CHECK_NR(hipFree(SortedPosY));
	CUDA_CHECK_NR(hipFree(VelX));
	CUDA_CHECK_NR(hipFree(SortedVelX));
	CUDA_CHECK_NR(hipFree(VelY));
	CUDA_CHECK_NR(hipFree(SortedVelY));
	CUDA_CHECK_NR(hipFree(ForceX));
	CUDA_CHECK_NR(hipFree(SortedForceX));
	CUDA_CHECK_NR(hipFree(ForceY));
	CUDA_CHECK_NR(hipFree(SortedForceY));
	CUDA_CHECK_NR(hipFree(Mass));
	CUDA_CHECK_NR(hipFree(Color));
}

__global__ void RandomParticlesKernel(float *posX, float *posY, float *velX, float *velY, float *mass, glm::vec4 *color, size_t count, float radius, glm::ivec2 dim, uint64_t seed)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	hiprandState state;
	hiprand_init(seed, idx, 0, &state);

	float sqrt = sqrtf(count);
	int gridWidth = sqrt;
	int gridHeight = ceilf(sqrt);
	float cellWidth = (float)dim.x / gridWidth;
	float cellHeight = (float)dim.y / gridHeight;

	for (; idx < count; idx += stride)
	{
		int row = idx / gridWidth;
		int col = idx % gridWidth;
		int baseX = col * cellWidth;
		int baseY = row * cellHeight;

		posX[idx] = baseX + hiprand_uniform(&state) * (cellWidth - 2 * radius) + radius;
		posY[idx] = baseY + hiprand_uniform(&state) * (cellHeight - 2 * radius) + radius;
		velX[idx] = hiprand_uniform(&state) * 2 * Config::RAND_PARTICLE_VELOCITY_MAX - Config::RAND_PARTICLE_VELOCITY_MAX;
		velY[idx] = hiprand_uniform(&state) * 2 * Config::RAND_PARTICLE_VELOCITY_MAX - Config::RAND_PARTICLE_VELOCITY_MAX;
		float t = hiprand_uniform(&state);
		mass[idx] = t * Config::PARTICLE_MASS_MAX + (1.0 - t) * Config::PARTICLE_MASS_MIN;
		color[idx] = glm::vec4(hiprand_uniform(&state), hiprand_uniform(&state), hiprand_uniform(&state), 1.0f);
	}
}

CudaParticleSystem *CudaParticleSystem::CreateRandom(uint32_t count, float radius, glm::ivec2 dim, ParticleSolver *solver)
{
	hipError_t cudaStatus;
	CudaParticleSystem *p = new CudaParticleSystem(count, radius, solver);
	time_t seed = time(NULL);

	RandomParticlesKernel << <BLOCKS_PER_GRID(count), THREADS_PER_BLOCK >> > (p->PosX, p->PosY, p->VelX, p->VelY, p->Mass, p->Color, count, radius, dim, seed);
	hipDeviceSynchronize();
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		Log::Error("CUDA Error " + std::to_string(cudaStatus) + ": " + hipGetErrorString(cudaStatus) + ". In file '" + __FILE__ + "' on line " + std::to_string(__LINE__));
		return nullptr;
	}

	return p;
}

CudaParticleSystem *CudaParticleSystem::CreateCircle(uint32_t count, float radius, glm::ivec2 dim, ParticleSolver *solver)
{
	CpuParticleSystem *p = CpuParticleSystem::CreateCircle(count, radius, dim, nullptr);
	CudaParticleSystem *pGPU = new CudaParticleSystem(count, radius, solver);
	hipMemcpy(pGPU->PosX, p->PosX, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->PosY, p->PosY, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->VelX, p->VelX, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->VelY, p->VelY, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->Mass, p->Mass, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->Color, p->Color, count * sizeof(glm::vec4), hipMemcpyHostToDevice);
	pGPU->Count = p->Count;
	delete p;
	return pGPU;
}

CudaParticleSystem *CudaParticleSystem::CreateBox(uint32_t count, float radius, glm::ivec2 dim, ParticleSolver *solver)
{
	CpuParticleSystem *p = CpuParticleSystem::CreateBox(count, radius, dim, nullptr);
	CudaParticleSystem *pGPU = new CudaParticleSystem(count, radius, solver);
	hipMemcpy(pGPU->PosX, p->PosX, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->PosY, p->PosY, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->VelX, p->VelX, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->VelY, p->VelY, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->Mass, p->Mass, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->Color, p->Color, count * sizeof(glm::vec4), hipMemcpyHostToDevice);
	pGPU->Count = p->Count;
	delete p;
	return pGPU;
}

CudaParticleSystem *CudaParticleSystem::CreateWaterfall(uint32_t count, float radius, glm::ivec2 dim, float velocity, int rows, ParticleSolver *solver)
{
	CpuParticleSystem *p = CpuParticleSystem::CreateWaterfall(count, radius, dim, velocity, rows, nullptr);
	CudaParticleSystem *pGPU = new CudaParticleSystem(count, radius, solver);
	hipMemcpy(pGPU->PosX, p->PosX, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->PosY, p->PosY, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->VelX, p->VelX, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->VelY, p->VelY, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->Mass, p->Mass, count * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->Color, p->Color, count * sizeof(glm::vec4), hipMemcpyHostToDevice);
	pGPU->Count = p->Count;
	delete p;
	return pGPU;
}

CudaParticleSystem *CudaParticleSystem::CreateFromCPU(CpuParticleSystem *p, ParticleSolver *solver)
{
	CudaParticleSystem *pGPU = new CudaParticleSystem(p->TotalCount, p->Radius, solver);
	hipMemcpy(pGPU->PosX, p->PosX, p->TotalCount * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->PosY, p->PosY, p->TotalCount * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->VelX, p->VelX, p->TotalCount * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->VelY, p->VelY, p->TotalCount * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->Mass, p->Mass, p->TotalCount * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(pGPU->Color, p->Color, p->TotalCount * sizeof(glm::vec4), hipMemcpyHostToDevice);
	pGPU->Count = p->Count;
	return pGPU;
}
