#include "hip/hip_runtime.h"
#include "cuda/CudaInstancedParticles.h"
#include "cpu/CpuInstancedParticles.h"
#include "utils/cuda_helper.h"

#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>

CudaInstancedParticles::CudaInstancedParticles(ParticleSystem *p, uint32_t ShaderProgram)
	: InstancedParticles(p, ShaderProgram)
{
	// Register buffer with CUDA
	CUDA_CHECK_NR(hipGraphicsGLRegisterBuffer(&m_CudaVBOResource, m_InstanceVBO, cudaGraphicsMapFlagsWriteDiscard));
}

CudaInstancedParticles::~CudaInstancedParticles()
{
	hipGraphicsUnregisterResource(m_CudaVBOResource);
}

__global__ void UpdateInstanceDataKernel(float *vboPtr, float *PosX, float *PosY, float2 Scale, float4 *Color, bool RandomColor, float4 StillColor, size_t count)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (; tid < count; tid += stride)
	{
		int vboIdx = 8 * tid;
		// Update position
		*(float2 *)(&vboPtr[vboIdx]) = make_float2(__ldg(&PosX[tid]), __ldg(&PosY[tid]));
		// Update scale
		*(float2 *)(&vboPtr[vboIdx + 2]) = Scale;
		// Update color
		*(float4 *)(&vboPtr[vboIdx + 4]) = RandomColor ? __ldg(&Color[tid]) : StillColor;
	}
}

void CudaInstancedParticles::UpdateParticleInstances()
{
	UpdateGraphicsData();
	if (!m_ParticleData.Count) return;

	// Map OpenGL buffer for writing from CUDA
	float *dPtr;
	size_t numBytes;
	CUDA_CHECK_NR(hipGraphicsMapResources(1, &m_CudaVBOResource));
	CUDA_CHECK_NR(hipGraphicsResourceGetMappedPointer((void **)&dPtr, &numBytes, m_CudaVBOResource));

	// Launch kernel to update instance data
	UpdateInstanceDataKernel << <BLOCKS_PER_GRID(m_ParticleData.Count), THREADS_PER_BLOCK >> > (
		dPtr,
		m_ParticleData.PosX,
		m_ParticleData.PosY,
		m_ParticleData.Scale,
		m_ParticleData.Color,
		m_ParticleData.RandomColor,
		m_ParticleData.StillColor,
		m_ParticleData.Count);
	hipDeviceSynchronize();
	CUDA_CHECK_NR(hipGetLastError());

	// Unmap buffer
	CUDA_CHECK_NR(hipGraphicsUnmapResources(1, &m_CudaVBOResource));
}
