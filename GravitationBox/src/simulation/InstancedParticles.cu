#include "hip/hip_runtime.h"
#include "InstancedParticles.h"
#include "glad/gl.h"
#include "cuda/cuda_helper.h"

#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>

__global__ void UpdateInstanceDataKernel(float *vboPtr, float *PosX, float *PosY, float2 Scale, float4 *Color, bool RandomColor, float4 StillColor, size_t count)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
	for (; tid < count; tid += stride)
	{
		int vboIdx = 8 * tid;
		// Update position
		*(float2 *)(&vboPtr[vboIdx]) = make_float2(__ldg(&PosX[tid]), __ldg(&PosY[tid]));
		// Update scale
		*(float2 *)(&vboPtr[vboIdx + 2]) = Scale;
		// Update color
		*(float4 *)(&vboPtr[vboIdx + 4]) = RandomColor ? __ldg(&Color[tid]) : StillColor;
	}
}

hipError_t InstancedParticles::UpdateParticleInstancesCUDA()
{
	if (!m_ParticleData.Count) return hipSuccess;

	UpdateGraphicsData();

	// Map OpenGL buffer for writing from CUDA
	float *dPtr;
	size_t numBytes;
	CUDA_CHECK(hipGraphicsMapResources(1, &m_CudaVBOResource));
	CUDA_CHECK(hipGraphicsResourceGetMappedPointer((void **)&dPtr, &numBytes, m_CudaVBOResource));

	// Launch kernel to update instance data
	UpdateInstanceDataKernel << <BLOCKS_PER_GRID(m_ParticleData.Count), THREADS_PER_BLOCK >> > (
		dPtr,
		m_ParticleData.PosX,
		m_ParticleData.PosY,
		m_ParticleData.Scale,
		m_ParticleData.Color,
		m_ParticleData.RandomColor,
		m_ParticleData.StillColor,
		m_ParticleData.Count);
	hipDeviceSynchronize();
	CUDA_CHECK(hipGetLastError());

	// Unmap buffer
	CUDA_CHECK(hipGraphicsUnmapResources(1, &m_CudaVBOResource));

	return hipSuccess;
}
